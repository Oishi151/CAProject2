#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>
using namespace std;

// Macro definitions for CUDA error checking
#define CUDA_CHECK_ERROR
#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)

// Error handling for CUDA calls
inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_CHECK_ERROR
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif
}

inline void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_CHECK_ERROR
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s.\n", file, line, hipGetErrorString(err));
        exit(-1);
    }
    err = hipDeviceSynchronize();
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n", file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif
}

// Function to generate random array of integers
int *makeRandArray(const int size, const int seed)
{
    srand(seed);
    int *array = new int[size];
    for (int i = 0; i < size; ++i)
    {
        array[i] = rand() % 100000; // Random integers between 0 and 99,999
    }
    return array;
}

// CUDA Kernel for parallel merge sort
const int MAX_THREADS_PER_BLOCK = 1024;

__global__ void mergeSort(int *array, int *temp, int size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure tid is within bounds
    if (tid >= size)
        return;

    // Iterative merge sort
    for (int width = 1; width < size; width *= 2)
    {
        int left = tid * 2 * width;
        int mid = min(left + width - 1, size - 1);
        int right = min(left + 2 * width - 1, size - 1);

        // Check for valid ranges
        if (left >= size)
            return;

        int i = left, j = mid + 1, k = left;

        // Merge the two halves
        while (i <= mid && j <= right)
        {
            if (array[i] <= array[j])
            {
                temp[k++] = array[i++];
            }
            else
            {
                temp[k++] = array[j++];
            }
        }

        while (i <= mid)
        {
            temp[k++] = array[i++];
        }

        while (j <= right)
        {
            temp[k++] = array[j++];
        }

        for (i = left; i <= right; i++)
        {
            array[i] = temp[i];
        }

        __syncthreads();
    }
}

int main(int argc, char *argv[])
{
    int size, seed;
    if (argc < 3)
    {
        std::cerr << "usage: "
                  << argv[0]
                  << " [amount of random nums to generate] [seed value for rand]"
                  << std::endl;
        exit(-1);
    }
    // convert cstrings to ints
    {
        std::stringstream ss1(argv[1]);
        ss1 >> size;
    }

    {
        std::stringstream ss1(argv[2]);
        ss1 >> seed;
    }

    cout << "Running merge sort for array size: " << size << endl;

    int *array = makeRandArray(size, seed); // Generate random array

    int *d_array, *d_temp;
    int numBlocks = (size + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

    // Allocate memory on GPU
    CudaSafeCall(hipMalloc((void **)&d_array, size * sizeof(int)));
    CudaSafeCall(hipMalloc((void **)&d_temp, size * sizeof(int)));

    // Copy data to GPU
    CudaSafeCall(hipMemcpy(d_array, array, size * sizeof(int), hipMemcpyHostToDevice));
    CudaSafeCall(hipDeviceSynchronize());
    // Timer setup
    hipEvent_t startTotal, stopTotal;
    float timeTotal;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventRecord(startTotal, 0);

    // Launch mergeSort kernel
    mergeSort<<<numBlocks, MAX_THREADS_PER_BLOCK>>>(d_array, d_temp, size);
    hipDeviceSynchronize();
    CudaCheckError();

    // Timer stop
    hipEventRecord(stopTotal, 0);
    hipEventSynchronize(stopTotal);
    hipEventElapsedTime(&timeTotal, startTotal, stopTotal);
    hipEventDestroy(startTotal);
    hipEventDestroy(stopTotal);

    cerr << "Total time in seconds for size " << size << ": " << timeTotal / 1000.0 << endl;

    // Copy sorted array back to host
    CudaSafeCall(hipMemcpy(array, d_array, size * sizeof(int), hipMemcpyDeviceToHost));
    // Free allocated memory
    delete[] array;
    hipFree(d_array);
    hipFree(d_temp);
    return 0;
}
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

// Function to generate random numbers
void generate_random_numbers(int *array, int num_elements, int seed) {
    srand(seed);
    for (int i = 0; i < num_elements; ++i) {
        array[i] = rand();
    }
}

// CUDA kernel for sorting each bucket using Thrust
__global__ void sort_buckets(int *buckets, int *bucket_offsets, int num_buckets, int bucket_size) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;

    // Determine the start and end index for the current bucket
    int start_idx = bucket_offsets[bid];
    int end_idx = (bid == num_buckets - 1) ? bucket_size : bucket_offsets[bid + 1];

    // Sort the bucket using Bubble Sort (You can replace this with Thrust sort for better performance)
    for (int i = start_idx; i < end_idx - 1; ++i) {
        for (int j = start_idx; j < end_idx - 1 - (i - start_idx); ++j) {
            if (buckets[j] > buckets[j + 1]) {
                int temp = buckets[j];
                buckets[j] = buckets[j + 1];
                buckets[j + 1] = temp;
            }
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " [number of random integers to generate] [seed value for random number generation]\n";
        return 1;
    }

    int num_elements = std::atoi(argv[1]);
    int seed = std::atoi(argv[2]);

    // Generate random numbers
    int *host_data = new int[num_elements];
    generate_random_numbers(host_data, num_elements, seed);

    // Allocate memory on device
    int *device_data;
    hipMalloc(&device_data, num_elements * sizeof(int));

    // Transfer data to device
    hipMemcpy(device_data, host_data, num_elements * sizeof(int), hipMemcpyHostToDevice);

    // Determine the number of threads and blocks based on the requirement
    int threads_per_block = 256;  // You can adjust this value based on your requirement
    int num_buckets = 1024;       // Number of buckets (You can adjust this value based on your requirement)
    int bucket_size = (num_elements + num_buckets - 1) / num_buckets;

    // Allocate memory for bucket offsets
    int *host_bucket_offsets = new int[num_buckets];
    int *device_bucket_offsets;
    hipMalloc(&device_bucket_offsets, num_buckets * sizeof(int));

    // Initialize bucket offsets
    for (int i = 0; i < num_buckets; ++i) {
        host_bucket_offsets[i] = i * bucket_size;
    }

    // Transfer bucket offsets to device
    hipMemcpy(device_bucket_offsets, host_bucket_offsets, num_buckets * sizeof(int), hipMemcpyHostToDevice);

    //std::cout << "Number of threads per block: " << threads_per_block << std::endl;
    //std::cout << "Number of buckets: " << num_buckets << std::endl;
    std::cout << "Total number of threads: " << threads_per_block * num_buckets << std::endl;

    /***********************************
     *
     create a cuda timer to time execution
     **********************************/
    hipEvent_t startTotal, stopTotal;
    float timeTotal;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventRecord(startTotal, 0);
    /***********************************
     *
     end of cuda timer creation
     **********************************/

    // Sort each bucket using CUDA kernel
    sort_buckets<<<num_buckets, threads_per_block>>>(device_data, device_bucket_offsets, num_buckets, num_elements);

    /***********************************
     *
     Stop and destroy the cuda timer
     **********************************/
    hipEventRecord(stopTotal, 0);
    hipEventSynchronize(stopTotal);
    hipEventElapsedTime(&timeTotal, startTotal, stopTotal);
    hipEventDestroy(startTotal);
    hipEventDestroy(stopTotal);
    /***********************************
     *
     end of cuda timer destruction
     **********************************/
    std::cerr << "Total time in seconds: " << timeTotal / 1000.0 << std::endl;

    // Transfer data back to host
    hipMemcpy(host_data, device_data, num_elements * sizeof(int), hipMemcpyDeviceToHost);

    // Print sorted numbers (commented out for large arrays)
    // for (int i = 0; i < num_elements; ++i) {
    //     std::cout << host_data[i] << " ";
    // }
    std::cout << std::endl;

    delete[] host_data;
    delete[] host_bucket_offsets;
    hipFree(device_data);
    hipFree(device_bucket_offsets);
    return 0;
}
